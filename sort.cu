/*
 * William Charles Grove
 * Bitonic sort array
 * Sorting Network Implementation
 *
 * 12/1/2016
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <limits.h>

int generate_int() {
	return rand() / 1000000;
}

void print_values(int *values, int length) {
	int j = length / 16;
	int remainder = length - (j * 16);
	int block = 0;
	for (int i = 0; i < j; ++i) {
		printf("\nBlock %d: ", block++);
		for (int index = 0; index < 16; index++) {
			printf("%d ", values[(i * 16) + index]);
		}

	}

	if (remainder > 0) {

		printf("\nBlock %d: ", block);

		for (int i = j * 16; i < (j * 16) + remainder; i++) {
			printf("%d ", values[i]);
		}

	}

}

__global__ void sn_iteration(int *value_array, int j, int k) {
	unsigned int i, x;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	x = i ^ j;

	/* The threads with the lowest ids sort the array. */
	if ((x) > i) {
		if ((i & k) == 0) {
			//Sort ascending
			if (value_array[i] > value_array[x]) {
				//swap
				float temp = value_array[i];
				value_array[i] = value_array[x];
				value_array[x] = temp;
			}
		}
		if ((i & k) != 0) {
			// Sort descending
			if (value_array[i] < value_array[x]) {
				// swap
				float temp = value_array[i];
				value_array[i] = value_array[x];
				value_array[x] = temp;
			}
		}
	}
}


//Sorting network method, activates  a kernel
void sorting_network(int *values, int block_num, int thread_num, int num_vals) {
	int *value_array;
	size_t size = num_vals * sizeof(int);

	hipMalloc((void**) &value_array, size);
	hipMemcpy(value_array, values, size, hipMemcpyHostToDevice);

	/*Unlike the Voronoi diagram,
	 * this process can be modeled with
	 * a one dimensional array.
	 */
	dim3 blocks(block_num, 1);
	dim3 threads(thread_num, 1);

	int j, k;
	/* Major step */
	for (k = 2; k <= num_vals; k <<= 1) {
		/* Minor step */
		for (j = k >> 1; j > 0; j = j >> 1) {
			sn_iteration<<<blocks, threads>>>(value_array, j, k);
		}
	}
	hipMemcpy(values, value_array, size, hipMemcpyDeviceToHost);
	hipFree(value_array);
}

int realloc_array(int **arr, int size) {

	int *temp;

	int fsize = size * sizeof(int);

	temp = (int*) realloc(*arr, fsize);

	if (temp == NULL) {
		printf("Error in realloc!");
		return 0;
	}

	*arr = temp;

	return 1;

}

int main(int argc, char* argv[]) {

	int *values = (int*) malloc(1 * sizeof(int));
	int num_vals = 0;

	if (argc == 1) {
		printf("There were no values passed into the function.");
	}

	if (argc == 2) {

		if ((atoi(argv[1]) == 0)) {
			//It is a file name

			FILE* file = fopen(argv[1], "r");

			if (file == NULL) {
				printf("Error opening file.\n");
				return -1;
			}

			int size = 1;

			int i = 0;
			int index = -1;

			while (!feof(file)) {
				fscanf(file, "%d", &i);

				index++;
				printf("\n[%d:%d]", index, i);
				values[index] = i;

				if (index + 1 == size) {
					size *= 2;
					if (realloc_array(&values, size) == 0) {
						return -1;
					}
				}
			}
			fclose(file);

			if (realloc_array(&values, index) == 0) {
				return -1;
			}

			num_vals = index;

		} else {
			//There is a number of values to generate

			num_vals = atoi(argv[1]);

			if (realloc_array(&values, num_vals) == 0) {
				return -1;
			}

			int i;
			for (i = 0; i < num_vals; ++i) {
				values[i] = generate_int();

			}
		}
	} else {
		//There is a list of values to read in

		for (int i = 1; i < argc; i++) {

			realloc_array(&values, i);
			values[i - 1] = atoi(argv[i]);
		}

		num_vals = argc - 1;

	}

	int padding = 0;
	int block_num = num_vals / 16;
	int remainder = num_vals % 16;

	if(block_num == 0){
		block_num = 1;
	}

	if (remainder > 0) {
		block_num++;
		padding = 16 - remainder;
		realloc_array(&values, num_vals + padding);

		for (int i = num_vals; i < num_vals + padding; i++) {
			values[i] = 50;
		}
	}

	for (int i = 0; i < num_vals + padding; i++) {
		printf("%d\n", values[i]);
	}

	printf("num_val: %d\n", num_vals);
	printf("padding: %d\n", padding);
	printf("remainder: %d\n", remainder);
	printf("block num: %d\n", block_num);
	printf("size of array: %d\n", sizeof(values)/sizeof(int));

	print_values(values, num_vals);

	sorting_network(values, block_num, block_num * 16, num_vals + padding);

	print_values(values, num_vals);
}
